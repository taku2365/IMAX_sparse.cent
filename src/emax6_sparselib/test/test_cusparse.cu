#include <hipsparse.h>
#include <hip/hip_runtime.h>

#include <iostream>
#include <random>
#include <unordered_map>
#include <map>
#include <set>
#include <time.h>
#include <sys/times.h>
#include <unistd.h>



inline const char* GetCuSparseErrorString(int error) {
    switch (error) {
    case HIPSPARSE_STATUS_NOT_INITIALIZED:
      return "CUSPARSE_STATUS_NOT_INITIALIZED";
    case HIPSPARSE_STATUS_ALLOC_FAILED:
      return "CUSPARSE_STATUS_ALLOC_FAILED";
    case HIPSPARSE_STATUS_INVALID_VALUE:
      return "CUSPARSE_STATUS_INVALID_VALUE";
    case HIPSPARSE_STATUS_ARCH_MISMATCH:
      return "CUSPARSE_STATUS_ARCH_MISMATCH";
    case HIPSPARSE_STATUS_MAPPING_ERROR:
      return "CUSPARSE_STATUS_MAPPING_ERROR";
    case HIPSPARSE_STATUS_EXECUTION_FAILED:
      return "CUSPARSE_STATUS_EXECUTION_FAILED";
    case HIPSPARSE_STATUS_INTERNAL_ERROR:
      return "CUSPARSE_STATUS_INTERNAL_ERROR";
    case HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED:
      return "CUSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED";
    case HIPSPARSE_STATUS_ZERO_PIVOT:
      return "CUSPARSE_STATUS_ZERO_PIVOT";
    }
    return "Unrecognized error";
  }
  
#ifndef CHECK_CUSPARSE_ERROR
#define CHECK_CUSPARSE_ERROR(fn)                  \
do {                                          \
    int error = static_cast<int>(fn);                      \
    if (error != HIPSPARSE_STATUS_SUCCESS) \
      std::cout << __LINE__ << " CUSPARSE: " << error << " " << GetCuSparseErrorString(error) << std::endl;\
} while (0)  // ; intentionally left off.
#endif  // CHECK_CUSPARSE_ERROR
  

struct GpuTimer
{
      hipEvent_t start;
      hipEvent_t stop;

      GpuTimer()
      {
            hipEventCreate(&start);
            hipEventCreate(&stop);
      }

      ~GpuTimer()
      {
            hipEventDestroy(start);
            hipEventDestroy(stop);
      }

      void Start()
      {
            hipEventRecord(start, 0);
      }

      void Stop()
      {
            hipEventRecord(stop, 0);
      }

      float Elapsed()
      {
            float elapsed;
            hipEventSynchronize(stop);
            hipEventElapsedTime(&elapsed, start, stop);
            return elapsed;
      }
};

unsigned long long     nanosec_sav, nanosec;
double  tmssave, tms;
long    ticksave, ticks;

void reset_nanosec()
{
  struct timespec ts;
  clock_gettime(0, &ts); /*CLOCK_REALTIME*/
  nanosec_sav = 1000000000*ts.tv_sec + ts.tv_nsec;
}

void show_nanosec()
{
  struct timespec ts;
  clock_gettime(0, &ts); /*CLOCK_REALTIME*/
  nanosec = 1000000000*ts.tv_sec + ts.tv_nsec;
  printf("nanosec: ARM:%llu\n", nanosec - nanosec_sav);
  nanosec_sav = nanosec;
}

void reset_time(void)
{
  struct tms    utms;

  times(&utms);
  ticksave = utms.tms_utime;
}

void show_time(void)
{
  struct tms    utms;

  times(&utms);
  ticks = utms.tms_utime;
  printf("====TOTAL-CPUS-TIME(w/o IO) %g sec===\n", (double)(ticks-ticksave)/sysconf(_SC_CLK_TCK));
  ticksave = ticks;
}

float get_random()
{
    static std::default_random_engine e;
    static std::uniform_real_distribution<> dis(0, 1); // rage 0 - 1
    return dis(e);
}

int get_random_int(int max) {
    static std::default_random_engine e;
    std::uniform_int_distribution<int> dis(0, max);
    return dis(e);
}

void generate_candidate_blocks(int R, int C, int BS_R, int BS_C, int num_blocks, int *weight_indptr, int *weight_indices) {
    std::map<int, std::set<int>> blocks;
    int num_r_block = R / BS_R;
    int num_c_block = C / BS_C;
    int curr_size = 0;
    while (curr_size < num_blocks) {
        int r = get_random_int(num_r_block);
        int c = get_random_int(num_c_block);
        if (blocks[r].count(c) == 0) {
          blocks[r].insert(c);
          curr_size++;
        }
    }

    int current_ptr = 0;
    int i;
    for (i = 0; i < num_r_block; i++) {
        weight_indptr[i] = current_ptr;
        for (auto block : blocks[i]) {
          weight_indices[current_ptr++] = block;  
        }
    }
    weight_indptr[i] = current_ptr;
}
hipsparseHandle_t handle{nullptr};

void test(int M, int N, int K, int BS_R, int BS_C, float sparsity) {

    float *data;
    float *weight;
    int *weight_ind;
    int *weight_ptr;
    int nnz = int((1-sparsity) * M * N);
    int num_blocks = int(nnz / (BS_R * BS_C)) + 1;

    data = (float*)malloc(M*K*sizeof(float));
    weight = (float*)malloc(num_blocks*BS_R*BS_C*sizeof(float));
    weight_ind = (int*)malloc(num_blocks*sizeof(int));
    weight_ptr = (int*)malloc((N+1)*sizeof(int));

    for (int i = 0; i < M*K; i++) {
        data[i] = get_random();
    }

    for (int i = 0; i < num_blocks*BS_R*BS_C; i++) {
        weight[i] = get_random();
    }

    generate_candidate_blocks(N, K, BS_R, BS_C, num_blocks, weight_ptr, weight_ind);


    float *data_cuda;
    float *weight_cuda;
    int *weight_ind_cuda;
    int *weight_ptr_cuda;
    float *output_cuda;
    
    float* output = (float*)malloc(M*N*sizeof(float));
    hipMalloc(&data_cuda, M*K*sizeof(float));
    hipMalloc(&weight_cuda, K*N*sizeof(float));
    hipMalloc(&weight_ind_cuda, num_blocks*sizeof(int));
    hipMalloc(&weight_ptr_cuda, (N+1)*sizeof(int));
    hipMalloc(&output_cuda, M*N*sizeof(float));
    

    hipMemcpy(data_cuda, data, M*K*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(weight_cuda, weight, num_blocks*BS_R*BS_C*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(weight_ind_cuda, weight_ind, num_blocks*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(weight_ptr_cuda, weight_ptr, (N+1)*sizeof(int), hipMemcpyHostToDevice);

    hipsparseMatDescr_t descr{nullptr};

    CHECK_CUSPARSE_ERROR(hipsparseCreateMatDescr(&descr));
    CHECK_CUSPARSE_ERROR(hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL));
    CHECK_CUSPARSE_ERROR(hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO));

    // A: sparse NxK
    // B: dense (transpose)MxK
    float alpha = 1.0;
    float beta = 0.0;

    GpuTimer timer;
    int n_runs =200;
    timer.Start();
    for (int i = 0; i < n_runs; i++) {
    hipMemcpy(data_cuda, data, M*K*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(weight_cuda, weight, num_blocks*BS_R*BS_C*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(weight_ind_cuda, weight_ind, num_blocks*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(weight_ptr_cuda, weight_ptr, (N+1)*sizeof(int), hipMemcpyHostToDevice);
    CHECK_CUSPARSE_ERROR(
      hipsparseSbsrmm(handle,
               HIPSPARSE_DIRECTION_ROW,
               HIPSPARSE_OPERATION_NON_TRANSPOSE,
               HIPSPARSE_OPERATION_NON_TRANSPOSE,
               N/BS_R, M, K/BS_C, num_blocks, &alpha,
               descr, weight_cuda, weight_ptr_cuda, weight_ind_cuda, BS_R,
               data_cuda, K,
               &beta, output_cuda, N));
    hipMemcpy( output, output_cuda, M*N*sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    }
    
    timer.Stop();
    std::cout << "M = " << M << " N = " << N << " K = " << K << " BS_R " << BS_R << " BS_C " << BS_C << " sparsity " << sparsity << std::endl;
    std::cout << timer.Elapsed() / n_runs << " ms" << std::endl;
}


int main() {
  CHECK_CUSPARSE_ERROR(hipsparseCreate(&handle));
  // if (argc != 2) {
  //     printf("arg fail\n");
  //     exit(0);
  // }
  // float density = atof(argv[1]);
  for (int M : {1024})
  for (int N : {1024}) 
  for (int K : {1024}) 
  for (int BS_R : {64,32}) 
  for (float sparsity : {0.0,0.1,0.2,0.3,0.4,0.5,0.6,0.7,0.8,0.85,0.9,0.95}) 
  test(M, N, K, BS_R, BS_R, sparsity);

  return 0;
}






