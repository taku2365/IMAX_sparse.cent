#include <hipsparse.h>
#include <hip/hip_runtime.h>

#include <iostream>
#include <random>
#include <unordered_map>
#include <map>
#include <set>



inline const char* GetCuSparseErrorString(int error) {
    switch (error) {
    case HIPSPARSE_STATUS_NOT_INITIALIZED:
      return "CUSPARSE_STATUS_NOT_INITIALIZED";
    case HIPSPARSE_STATUS_ALLOC_FAILED:
      return "CUSPARSE_STATUS_ALLOC_FAILED";
    case HIPSPARSE_STATUS_INVALID_VALUE:
      return "CUSPARSE_STATUS_INVALID_VALUE";
    case HIPSPARSE_STATUS_ARCH_MISMATCH:
      return "CUSPARSE_STATUS_ARCH_MISMATCH";
    case HIPSPARSE_STATUS_MAPPING_ERROR:
      return "CUSPARSE_STATUS_MAPPING_ERROR";
    case HIPSPARSE_STATUS_EXECUTION_FAILED:
      return "CUSPARSE_STATUS_EXECUTION_FAILED";
    case HIPSPARSE_STATUS_INTERNAL_ERROR:
      return "CUSPARSE_STATUS_INTERNAL_ERROR";
    case HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED:
      return "CUSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED";
    case HIPSPARSE_STATUS_ZERO_PIVOT:
      return "CUSPARSE_STATUS_ZERO_PIVOT";
    }
    return "Unrecognized error";
  }
  
#ifndef CHECK_CUSPARSE_ERROR
#define CHECK_CUSPARSE_ERROR(fn)                  \
do {                                          \
    int error = static_cast<int>(fn);                      \
    if (error != HIPSPARSE_STATUS_SUCCESS) \
      std::cout << __LINE__ << " CUSPARSE: " << error << " " << GetCuSparseErrorString(error) << std::endl;\
} while (0)  // ; intentionally left off.
#endif  // CHECK_CUSPARSE_ERROR
  

struct GpuTimer
{
      hipEvent_t start;
      hipEvent_t stop;

      GpuTimer()
      {
            hipEventCreate(&start);
            hipEventCreate(&stop);
      }

      ~GpuTimer()
      {
            hipEventDestroy(start);
            hipEventDestroy(stop);
      }

      void Start()
      {
            hipEventRecord(start, 0);
      }

      void Stop()
      {
            hipEventRecord(stop, 0);
      }

      float Elapsed()
      {
            float elapsed;
            hipEventSynchronize(stop);
            hipEventElapsedTime(&elapsed, start, stop);
            return elapsed;
      }
};


float get_random()
{
    static std::default_random_engine e;
    static std::uniform_real_distribution<> dis(0, 1); // rage 0 - 1
    return dis(e);
}

int get_random_int(int max) {
    static std::default_random_engine e;
    std::uniform_int_distribution<int> dis(0, max);
    return dis(e);
}

void generate_candidate_blocks(int R, int C, int BS_R, int BS_C, int num_blocks, int *weight_indptr, int *weight_indices) {
    std::map<int, std::set<int>> blocks;
    int num_r_block = R / BS_R;
    int num_c_block = C / BS_C;
    int curr_size = 0;
    while (curr_size < num_blocks) {
        int r = get_random_int(num_r_block);
        int c = get_random_int(num_c_block);
        if (blocks[r].count(c) == 0) {
          blocks[r].insert(c);
          curr_size++;
        }
    }

    int current_ptr = 0;
    int i;
    for (i = 0; i < num_r_block; i++) {
        weight_indptr[i] = current_ptr;
        for (auto block : blocks[i]) {
          weight_indices[current_ptr++] = block;  
        }
    }
    weight_indptr[i] = current_ptr;
}
hipsparseHandle_t handle{nullptr};

void test(int M, int N, int K, int BS_R, int BS_C, float density) {

    float *data;
    float *weight;
    int *weight_ind;
    int *weight_ptr;
    int nnz = int(density * M * N);
    int num_blocks = int(nnz / (BS_R * BS_C)) + 1;

    data = (float*)malloc(M*K*sizeof(float));
    weight = (float*)malloc(num_blocks*BS_R*BS_C*sizeof(float));
    weight_ind = (int*)malloc(num_blocks*sizeof(int));
    weight_ptr = (int*)malloc((N+1)*sizeof(int));

    for (int i = 0; i < M*K; i++) {
        data[i] = get_random();
    }

    for (int i = 0; i < num_blocks*BS_R*BS_C; i++) {
        weight[i] = get_random();
    }

    generate_candidate_blocks(N, K, BS_R, BS_C, num_blocks, weight_ptr, weight_ind);


    float *data_cuda;
    float *weight_cuda;
    int *weight_ind_cuda;
    int *weight_ptr_cuda;
    float *output_cuda;
    
    hipMalloc(&data_cuda, M*K*sizeof(float));
    hipMalloc(&weight_cuda, K*N*sizeof(float));
    hipMalloc(&weight_ind_cuda, num_blocks*sizeof(int));
    hipMalloc(&weight_ptr_cuda, (N+1)*sizeof(int));
    hipMalloc(&output_cuda, M*N*sizeof(float));


    hipMemcpy(data_cuda, data, M*K*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(weight_cuda, weight, num_blocks*BS_R*BS_C*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(weight_ind_cuda, weight_ind, num_blocks*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(weight_ptr_cuda, weight_ptr, (N+1)*sizeof(int), hipMemcpyHostToDevice);

    hipsparseMatDescr_t descr{nullptr};

    CHECK_CUSPARSE_ERROR(hipsparseCreateMatDescr(&descr));
    CHECK_CUSPARSE_ERROR(hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL));
    CHECK_CUSPARSE_ERROR(hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO));

    // A: sparse NxK
    // B: dense (transpose)MxK
    float alpha = 1.0;
    float beta = 0.0;

    GpuTimer timer;
    int n_runs = 20;
    timer.Start();
    for (int i = 0; i < n_runs; i++) {
    CHECK_CUSPARSE_ERROR(
      hipsparseSbsrmm(handle,
               HIPSPARSE_DIRECTION_ROW,
               HIPSPARSE_OPERATION_NON_TRANSPOSE,
               HIPSPARSE_OPERATION_NON_TRANSPOSE,
               N/BS_R, M, K/BS_C, num_blocks, &alpha,
               descr, weight_cuda, weight_ptr_cuda, weight_ind_cuda, BS_R,
               data_cuda, K,
               &beta, output_cuda, N));
    }
    timer.Stop();
    std::cout << "M = " << M << " N = " << N << " K = " << K << " BS_R " << BS_R << " BS_C " << BS_C << " density " << density << std::endl;
    std::cout << timer.Elapsed() / n_runs << " ms" << std::endl;
}


int main() {
  CHECK_CUSPARSE_ERROR(hipsparseCreate(&handle));
  for (int M : {1, 8, 64})
  for (int N : {1024}) 
  for (int K : {1024}) 
  for (int BS_R : {8, 16, 32}) 
  for (float density : {0.20, 0.15, 0.05}) 
  test(M, N, K, BS_R, BS_R, density);

  return 0;
}