#include "hip/hip_runtime.h"
/*                                   疎行列演算withrtx3090*/
/*                               Primary writer: R.Funai */
/*                         funai.ryotaro.fs8@is.naist.jp */

#include <stdio.h>
#include <fcntl.h>
#include <math.h>
#include <time.h>
#include <cinttypes>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#include <unistd.h>
#include <sys/times.h>
#include <sys/mman.h>
#include <sys/resource.h>
#include <pthread.h>
#include <iostream>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>
#include <hipsparse.h>
#include <thrust/device_vector.h>

typedef unsigned char      Uchar;
typedef unsigned short     Ushort;
typedef unsigned int       Uint;
typedef unsigned long long Ull;
typedef long long int      Sll;

//-----------------
// Device Kernel
//-----------------
__global__ void Device_Kernel(void)
{
	float ix = threadIdx.x + blockIdx.x * blockDim.x;
	float iy = threadIdx.y + blockIdx.y * blockDim.y;
	printf("Device (ix, iy) = (%d, %d) : \
threadIdx.x = %d, blockIdx.x = %d, blockDim.x = %d, \
threadIdx.y = %d, blockIdx.y = %d, blockDim.y = %d\n",
	        ix, iy, 
	        threadIdx.x, blockIdx.x, blockDim.x,
	        threadIdx.y, blockIdx.y, blockDim.y);
}

/* LMM:16KB, RMM:64KB: M/NCHIP=124 M/NCHIP/RMGRP=31 */
/* A A   B B B B B B   C C C C C C */
/* A A   B B B B B B   C C C C C C */
/* A A                 C C C C C C */
/* A A                 C C C C C C */
/* M1=4 L=2, M2=6      L<M1,M2     */

#define M1 240LL
#define M2 240LL
#define L  240LL
#define LP 24LL
#define BLOCK_SIZE (32,32)
#define GRID_SIZE (8,8)

// Cuda実行時、ランタイムAPIをCHECKで囲むとエラーが見れるようにする
#if 0
#define CHECK(func) 
{
    {
        const hipError_t error = func; //funcで指定したAPIの戻り値を取得(error/success)
        // Errorなら以下を表示する
        if (error != hipSuccess)
        {
            printf("Error: %s:%d, ",__FILE__, __LINE__);    //FILEと行数
            printf("Code:%d, Reason: %s\n", error, hipGetErrorString(error));  //Error内容
            hipDeviceReset();  //デバイスのリセット
            exit(EXIT_FAILURE); //エラー終了
        }
    }
}
#endif

typedef struct {
  float d; /* low */
  Uint  x; /* high */
} packed;

float A[M1*M2], B[M1*M2], B_T[M1*M2], C_Orig[M1*M2], C_Gpu[M1*M2];
int row, col, n;
int top, blk;
int w, h;
int count0, count1, count2;

void orig();
void gpu();
void reset_nanosec();
void show_nanosec();
void reset_time();
void show_time();

int main(void){
   int row, col;

  /* 乗数行列の生成 */ 
  /**************************************************/
  /* A                                              */
  /**************************************************/
  for (row=0; row<M1; row++) {
    for (col=0; col<L; col++) {
      if (abs(row-col) < LP/2)
        A[row*L+col] = (float)(row-col)/(float)(L/(128/LP));
      else  A[row*L+col] = 0;
    }
  }

  /* 被乗数行列の生成 */ 
  /**************************************************/
  /* B                                              */
  /**************************************************/
  for (row=0; row<M1; row++) {
    for (col=0; col<M2; col++) {
      if (abs(row-col) < LP/2)
        B[row*L+col] = (float)(row-col)/(float)(L/(128/LP));
      else B[row*L+col] = 0;
    }
  }
  /* 転置行列 */
  for(row=0;row<M1;++row){
    for(col=0;col<M2;++col){
     B_T[col*L+row]=B[row*L+col];
    }
  }
  /*結果格納行列の生成*/
  /**************************************************/
  /* C                                              */
  /**************************************************/
  for (row=0; row<M2; row++) {
    for (col=0; col<L; col++) {
        C_Orig[row*L+col] = 0;
        C_Gpu[row*L+col] = 0;
    }
  }
  orig();
//RTX3090によるCuSparseの実行
  gpu();

// 検算
  for (row=0; row<M1; row++) {
    for (col=0; col<M2; col++) {
      if (C_Orig[row*M2+col] != C_Gpu[row*M2+col]) {
        count2++;
        printf("C_Orig[%d][%d]=%f C_Gpu[%d][%d]=%f\n", row, col, (double)C_Orig[row*M2+col],
                                                        row, col, (double)C_Gpu[row*M2+col]);
      }
    }
  }
  if (count2)
    printf("Num of diffs: %d\n", count2);
  else
    printf("Results are equal\n");

  return (0);
}

void orig() {
  printf("<<<ORIG>>>\n");
  reset_nanosec();
  for (row=0; row<M1; row++) {
    for (col=0; col<M2; col++) {
      for (n=0; n<L; n++) {
        if (n==0) *(float*)&C_Orig[row*M2+col]  = *(float*)&A[row*L+n] * *(float*)&B[col*L+n];
        else      *(float*)&C_Orig[row*M2+col] += *(float*)&A[row*L+n] * *(float*)&B[col*L+n];
        count0++;
      }
    }
  }
  show_nanosec();
}

// ここを書き換える
void gpu() {
  hipError_t cudaStat;
  hipblasStatus_t stat;
  hipblasHandle_t handle;
  float *dA,*dB,*dC;


  printf("<<<RTX3090>>>\n");
  
  // GPUのメモリを確保 
  cudaStat = hipMalloc((void**)&dA, M1*M2*sizeof(float));
  if (cudaStat != hipSuccess) printf ("device memory allocation failed");
  cudaStat = hipMalloc((void**)&dB, M1*M2*sizeof(float));
  if (cudaStat != hipSuccess) printf ("device memory allocation failed");
  cudaStat = hipMalloc((void**)&dC, M1*M2*sizeof(float));
  if (cudaStat != hipSuccess) printf ("device memory allocation failed");
  stat = hipblasCreate(&handle);
  hipblasSetMathMode(handle,HIPBLAS_TENSOR_OP_MATH);  
  
  // 行列のコピー
  if (stat != HIPBLAS_STATUS_SUCCESS) printf ("CUBLAS initialization failed\n");
  stat = hipblasSetMatrix (M1, M2, sizeof(float), A, M1, dA, M1);
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    printf ("data download failed");
    hipFree (dA);
    hipblasDestroy(handle); 
  }
  stat = hipblasSetMatrix (M1, M2, sizeof(float), B_T, M1, dB, M1);
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    printf ("data download failed");
    hipFree (dB);
    hipblasDestroy(handle); 
  }
  stat = hipblasSetMatrix (M1, M2, sizeof(float), C_Gpu, M1, dC, M1);
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    printf ("data download failed");
    hipFree (dC);
    hipblasDestroy(handle); 
  }

  // 行列演算の実行
  // y=α*Ax + β*yと書く必要があるため、alpha=1.0f, beta=0.0f
  /*https://qiita.com/t-tkd3a/items/098f51c2cc40a5bddc38
    hipblasSgemm(    
        handle,
        HIPBLAS_OP_N, //行列A 転置有無
        HIPBLAS_OP_N, //行列B 転置有無
        num,    // 行列Aの行数
        num,    // 行列Bの列数
        num,    // 行列Aの列数(=行列Ｂの行数)
        &alpha, // 行列の積に掛ける値(なければ1)
        devA,   // 行列A
        num,    // 行列Aの行数
        devB,   // 行列B
        num,    // 行列Bの行数
        &beta,  // 行列Cに掛けるスカラ値(なければ0)
        devC,   // 行列Cの初期値 兼 出力先
        num // 行列Cの行数
);*/
  reset_nanosec();
  float alpha=1.0f, beta=0.0f;
  hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,M1,M2,M2,&alpha,dA,M1,dB,M1,&beta,dC,M1);

  // 演算の終わりとともに計測終了
  show_nanosec();
  // 
  stat = hipblasGetMatrix(M1,M2,sizeof(float),dC,M2,C_Gpu,M2);
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    printf ("data upload failed");
    hipFree (dC);
    hipblasDestroy(handle);
  }
  hipFree(dA);
  hipFree(dB);
  hipFree(dC);
  hipblasDestroy(handle);
}

Ull     nanosec_sav, nanosec;
double  tmssave, tms;
long    ticksave, ticks;
struct  rusage rusage;

void reset_nanosec()
{
  struct timespec ts;
  clock_gettime(0, &ts); /*CLOCK_REALTIME*/
  nanosec_sav = 1000000000*ts.tv_sec + ts.tv_nsec;
}

void show_nanosec()
{
  struct timespec ts;
  clock_gettime(0, &ts); /*CLOCK_REALTIME*/
  nanosec = 1000000000*ts.tv_sec + ts.tv_nsec;
  printf("nanosec: ARM:%llu\n", nanosec - nanosec_sav);
  nanosec_sav = nanosec;
}

void reset_time(void)
{
  struct tms    utms;

  times(&utms);
  ticksave = utms.tms_utime;
}

void show_time(void)
{
  struct tms    utms;

  times(&utms);
  ticks = utms.tms_utime;
  printf("====TOTAL-CPUS-TIME(w/o IO) %g sec===\n", (double)(ticks-ticksave)/sysconf(_SC_CLK_TCK));
  ticksave = ticks;
}
